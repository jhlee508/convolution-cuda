#include <hipDNN.h>

#include <cstdio>
#include <cstdlib>

#include "convolution_cudnn.h"

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

#define CHECK_CUDNN(call)                                              \
  do {                                                                 \
    hipdnnStatus_t status_ = call;                                      \
    if (status_ != HIPDNN_STATUS_SUCCESS) {                             \
      fprintf(stderr, "CUDNN error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipdnnGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)


static hipdnnHandle_t handle;
static hipdnnTensorDescriptor_t input_desc;
static hipdnnFilterDescriptor_t filter_desc;
static hipdnnConvolutionDescriptor_t conv_desc;
static hipdnnTensorDescriptor_t output_desc;
static int ON, OC, OH, OW;
static float *I_gpu, *F_gpu, *O_gpu, *workspace;
static hipdnnConvolutionFwdAlgoPerf_t best_algo;

static const char *algo_to_string(hipdnnConvolutionFwdAlgo_t algo);

void convolution_cudnn_initialize(int N, int C, int H, int W, int K, int R,
                                  int S, int pad_h, int pad_w, int stride_h,
                                  int stride_w, int dilation_h,
                                  int dilation_w) {
  CHECK_CUDNN(hipdnnCreate(&handle));

  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT, N, C, H, W));

  CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filter_desc));
  CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT,
                                         HIPDNN_TENSOR_NCHW, K, C, R, S));

  CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(
      conv_desc, pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

  CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(
      conv_desc, input_desc, filter_desc, &ON, &OC, &OH, &OW));

  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT, ON, OC, OH, OW));

  /* Enable TC if available */
  CHECK_CUDNN(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));

  int max_algo_count;
  CHECK_CUDNN(
      cudnnGetConvolutionForwardAlgorithmMaxCount(handle, &max_algo_count));

  int returned_algo_count;
  hipdnnConvolutionFwdAlgoPerf_t algo_perfs[max_algo_count];
  CHECK_CUDNN(hipdnnFindConvolutionForwardAlgorithm(
      handle, input_desc, filter_desc, conv_desc, output_desc, max_algo_count,
      &returned_algo_count, algo_perfs));

  printf("%-50s %-12s %-15s %-28s %-25s\n",
    "Algorithm", "Time (sec)", "Memory (bytes)", "Status", "MathType");
  printf("------------------------------------------------------------------");
  printf("------------------------------------------------------------------\n");
 
  for (int i = 0; i < returned_algo_count; ++i) {
    printf("%-50s %-12.6f %-15lu %-28s %-25s\n", 
      algo_to_string(algo_perfs[i].algo),
      algo_perfs[i].time,
      algo_perfs[i].memory,
      hipdnnGetErrorString(algo_perfs[i].status),
      algo_perfs[i].mathType == HIPDNN_TENSOR_OP_MATH
        ? "HIPDNN_TENSOR_OP_MATH"
        : "HIPDNN_DEFAULT_MATH");
  }

  best_algo = algo_perfs[0];
  printf("Using algorithm: %s\n", algo_to_string(best_algo.algo));

  CHECK_CUDA(hipMalloc(&I_gpu, N * C * H * W * sizeof(float)));
  CHECK_CUDA(hipMalloc(&F_gpu, K * C * R * S * sizeof(float)));
  CHECK_CUDA(hipMalloc(&O_gpu, ON * OC * OH * OW * sizeof(float)));
  CHECK_CUDA(hipMalloc(&workspace, best_algo.memory));
}

void convolution_cudnn(float *I, float *F, float *O, int N, int C, int H, int W,
                       int K, int R, int S, int pad_h, int pad_w, int stride_h,
                       int stride_w, int dilation_h, int dilation_w) {
  CHECK_CUDA(hipMemcpy(I_gpu, I, N * C * H * W * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(F_gpu, F, K * C * R * S * sizeof(float),
                        hipMemcpyHostToDevice));

  const float alpha = 1, beta = 0;
  CHECK_CUDNN(hipdnnConvolutionForward(
      handle, &alpha, input_desc, I_gpu, filter_desc, F_gpu, conv_desc,
      best_algo.algo, workspace, best_algo.memory, &beta, output_desc, O_gpu));

  CHECK_CUDA(hipMemcpy(O, O_gpu, ON * OC * OH * OW * sizeof(float),
                        hipMemcpyDeviceToHost));
}

void convolution_cudnn_finalize(int N, int C, int H, int W, int K, int R, int S,
                                int pad_h, int pad_w, int stride_h,
                                int stride_w, int dilation_h, int dilation_w) {
  CHECK_CUDA(hipFree(I_gpu));
  CHECK_CUDA(hipFree(F_gpu));
  CHECK_CUDA(hipFree(O_gpu));
  CHECK_CUDA(hipFree(workspace));

  CHECK_CUDNN(hipdnnDestroyTensorDescriptor(input_desc));
  CHECK_CUDNN(hipdnnDestroyFilterDescriptor(filter_desc));
  CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CHECK_CUDNN(hipdnnDestroyTensorDescriptor(output_desc));
  CHECK_CUDNN(hipdnnDestroy(handle));
}

const char *algo_to_string(hipdnnConvolutionFwdAlgo_t algo) {
  switch (algo) {
    case HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM";
    case HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM";
    case HIPDNN_CONVOLUTION_FWD_ALGO_GEMM:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_GEMM";
    case HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT";
    case HIPDNN_CONVOLUTION_FWD_ALGO_FFT:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_FFT";
    case HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING";
    case HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD";
    case HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED";
    case HIPDNN_CONVOLUTION_FWD_ALGO_COUNT:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_COUNT";
    default: return "<unknown algorithm>";
  }
}